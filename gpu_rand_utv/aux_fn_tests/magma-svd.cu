#include "hip/hip_runtime.h"
/*
how to compile:
nvcc -I/usr/local/magma/include -c magma-svd.cu
nvcc -o magma-svd.x magma-svd.o -L/usr/local/magma/lib \
	-lmagma -lcusolver -lcublas -lgomp
*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <hipblas.h>
#include <hipsolver.h>

#include <magma.h>

#define min( a,b ) ( (a) > (b) ? (b) : (a) )
#define max( a,b ) ( (a) > (b) ? (a) : (b) )

static void gpu_print_double_matrix( char * name, int m_A, int n_A,  
               double * buff_A, int ldim_A ) { 
 int  i, j; 

 double * A_pc; 
 
 A_pc = ( double * ) malloc( m_A * n_A * sizeof( double ) ); 
 
 hipMemcpy( A_pc, buff_A, m_A * n_A * sizeof( double ), 
          hipMemcpyDeviceToHost ); 

   printf( "%s = [\n", name ); 
   for( i = 0; i < m_A; i++ ) { 
     for( j = 0; j < n_A; j++ ) { 
     printf( "%.16e ", A_pc[ i + j * ldim_A ] ); 
   } 
   printf( "\n" ); 
  } 
  printf( "];\n" ); 
  
  free( A_pc ); 
} 

static void print_double_matrix(const char * name, int m_A, int n_A, 
                double * buff_A, int ldim_A ) {
  int  i, j;

  printf( "%s = [\n", name );
  for( i = 0; i < m_A; i++ ) {
    for( j = 0; j < n_A; j++ ) {
      printf( "%le ", buff_A[ i + j * ldim_A ] );
    }
    printf( "\n" );
  }
  printf( "];\n" );
}

__global__ 
static void Normal_random_matrix_kern( int m_A, int n_A, 
				double * A_pg, int ldim_A,
				hiprandState_t state, 
				unsigned long long rand_seed ) {
  // fill matrix A with random numbers of standard normal distribution
 
  int ij = blockIdx.x * blockDim.x + threadIdx.x;
  
  int i = ij - m_A * ( ij / m_A ); // floor implicitly taken in div
  int j = ij / m_A; // floor implicitly taken
  
  //seed RNG
  hiprand_init( rand_seed, ij, 0, & state );

  if ( ij < m_A * n_A ) {
    A_pg[ i + j * ldim_A ] = hiprand_normal( & state );
  }

}

static void Normal_random_matrix( int m_A, int n_A,
				double * A_pg, int ldim_A,
				hiprandState_t state, 
				unsigned long long * rs_pt ) {
  // host function which fills the gpu array with random numbers
  // of standard normal distribution

  Normal_random_matrix_kern<<<m_A*n_A, 1 >>>( m_A, n_A, A_pg, ldim_A, state, * rs_pt );

  * rs_pt = * rs_pt + 1;

}

__global__
static void Set_ss_diag_mat_kern( int m_A, int n_A,
				double * A_pg, int ldim_A,
				double * ss_pg ) {
  // kernel function which sets matrix represented by A_pg to a diagonal
  // matrix with the svs of A on the diagonal

  int ij = blockIdx.x * blockDim.x + threadIdx.x;
  
  // determine column, row indices for current thread
  int i = ij - m_A * ( ij / m_A ); // floor implicitly taken in div
  int j = ij / m_A; // floor implicitly taken

  // fill in matrix
  if ( ( ij < m_A * n_A ) && ( i == j ) ) {
    A_pg[ i + j * ldim_A ] = ss_pg[ i ];
  }
  else if ( ij < m_A * n_A ) {
    A_pg[ i + j * ldim_A ] = 0.0; 
  }
  
}

static void Set_ss_diag_mat( int m_A, int n_A, double * A_pg, int ldim_A,
				double * ss_pg ) {
  // host function which sets matrix represented by A_pg to a diagonal
  // matrix with the svs of A on the diagonal
  Set_ss_diag_mat_kern<<< ( m_A * n_A / 256 ) + 1, 256 >>>( m_A, n_A, A_pg, ldim_A,
				ss_pg );
}

static void local_magsvd( int m_A, int n_A, double * A_pg, int ldim_A,
				double * ss_pg,
				double * U_pg, int ldim_U,
				double * Vt_pg, int ldim_Vt ) {
  // given an m_A x n_A matrix A stored in device memory in A_pg,
  // this function computes the svd on the device
  
  // declare and initialize auxiliary variables
  
  double * ss_p;
  double * A_p, * U_p, * Vt_p;
  
  double * work_p = NULL; // work buffer array
  int lwork = 0; // size of work buffer
  int * iwork_p; // an integer workspace array
  magma_int_t * magInfo = NULL; // stored in device
  
  // vars for determining size of work array
  int nb = magma_get_dgesvd_nb( m_A, n_A );
  int A_mx, A_mn;

  // get max and min
  A_mx = max( m_A, n_A );
  A_mn = min( m_A, n_A );

  // allocate space for devInfo on device
  magInfo = ( magma_int_t * ) malloc( sizeof( magma_int_t ) );

  // determine size of work array
  // all SVDs for randUTV will be square, which determines our
  // equation for finding lwork
  lwork = 3*A_mn + max( 3*A_mn*A_mn + 4*A_mn, (A_mx+A_mn)*nb );
  
  // now we can allocate memory for work arrays
  magma_dmalloc_pinned( & work_p, lwork );
  magma_imalloc_pinned( & iwork_p, 8 * A_mn );

  // arrays must be in host memory for magma svd
  A_p = ( double * ) malloc( m_A * n_A * sizeof( double ) ); 
  U_p = ( double * ) malloc( m_A * m_A * sizeof( double ) ); 
  Vt_p = ( double * ) malloc( n_A * n_A * sizeof( double ) ); 
  ss_p = ( double * ) malloc( A_mn * sizeof( double ) ); 
  
  hipMemcpy( A_p, A_pg, m_A * n_A * sizeof( double ),
				hipMemcpyDeviceToHost );
  hipMemcpy( U_p, U_pg, m_A * m_A * sizeof( double ),
				hipMemcpyDeviceToHost );
  hipMemcpy( Vt_p, Vt_pg, n_A * n_A * sizeof( double ),
				hipMemcpyDeviceToHost );

  // compute factorization
  magma_dgesdd( MagmaAllVec,
				m_A, n_A, A_p, ldim_A,
				ss_p, 
				U_p, ldim_U,
				Vt_p, ldim_Vt,
				work_p, lwork, iwork_p, magInfo );
  
  // transfer results back to device
  hipMemcpy( U_pg, U_p, m_A * m_A * sizeof( double ),
				hipMemcpyHostToDevice );
  hipMemcpy( Vt_pg, Vt_p, n_A * n_A * sizeof( double ),
				hipMemcpyHostToDevice );
  hipMemcpy( ss_pg, ss_p, A_mn * sizeof( double ),
				hipMemcpyHostToDevice );

  // set contents of A_pg to zeros with svs on diagonal
  Set_ss_diag_mat( m_A, n_A, A_pg, ldim_A, ss_pg );

  // free memory
  free( A_p );
  free( U_p );
  free( Vt_p );
  free( ss_p );
   
  free( magInfo );
  magma_free_pinned( work_p );
  magma_free_pinned( iwork_p );
  
}

static void gpu_dgemm( char opA, char opB, double alpha, 
				int m_A, int n_A, double * A_pg, int ldim_A,
				int m_B, int n_B, double * B_pg, int ldim_B,
				double beta,
				int m_C, int n_C, double * C_pg, int ldim_C ) {

  // generate the correct transpose option identifier that CUBLAS accepts
  // also determine the correct "middle" dim of the mult
  hipblasOperation_t cutransA, cutransB;
  int middle_dim;

  if ( opA == 'N' ) { cutransA = HIPBLAS_OP_N; middle_dim = n_A; }
  else if ( opA == 'T' ) { cutransA = HIPBLAS_OP_T; middle_dim = m_A; }

  if ( opB == 'N' ) { cutransB = HIPBLAS_OP_N; }
  else if ( opB == 'T' ) { cutransB = HIPBLAS_OP_T; }


  // create a handle for CUBLAS
  hipblasHandle_t handle;
  hipblasCreate( & handle );

  // do the multiplication
  hipblasDgemm( handle, cutransA, cutransB,
				m_C, n_C, middle_dim, & alpha,
				A_pg, ldim_A, B_pg, ldim_B, 
				& beta, C_pg, ldim_C );

  // destroy the handle
  hipblasDestroy( handle );

}




int main() {

  // declare, initialize variables
  int m_A, n_A, ldim_A;
  m_A = 300; n_A = 300; ldim_A = m_A;
  const char * A_name = "A";
  double * A_pc, * A_pg, * A_pgc;
  
  int i;
  double err = 0.0, A_norm = 0.0; // for checking factorization later

  char n = 'N', t = 'T';
  double d_one = 1.0, d_zero = 0.0, d_neg_one = -1.0;
  double * ss_pg;
  int m_U, n_U, ldim_U;
  m_U = m_A; n_U = m_U; ldim_U = m_U;
  double * U_pg;
  int m_Vt, n_Vt, ldim_Vt;
  m_Vt = n_A; n_Vt = m_Vt; ldim_Vt = m_Vt;
  double * Vt_pg;

  hiprandState_t state; // we store a random state for every thread
  unsigned long long rand_seed = 7;
  unsigned long long * rs_pt = & rand_seed;

  hipError_t cudaStat = hipSuccess;
 
  // initialize magma
  magma_init();

  // allocate array on host (cpu)
  A_pc = ( double * ) malloc( m_A * n_A * sizeof( double ) );

  // allocate array on device (gpu)
  cudaStat = hipMalloc( ( void ** ) & A_pg, m_A * n_A * sizeof( double ) );
  assert( cudaStat == hipSuccess );
 

  cudaStat = hipMalloc( ( void ** ) & A_pgc, m_A * n_A * sizeof( double ) );
  assert( cudaStat == hipSuccess );


  // allocate arrays for svd output
  cudaStat = hipMalloc( ( void ** ) & ss_pg, n_A * sizeof( double ) );
  assert( cudaStat == hipSuccess );
  
  cudaStat = hipMalloc( ( void ** ) & U_pg, m_U * n_U * sizeof( double ) );
  assert( cudaStat == hipSuccess );
  
  cudaStat = hipMalloc( ( void ** ) & Vt_pg, m_Vt * n_Vt * sizeof( double ) );
  assert( cudaStat == hipSuccess );

  // fill gpu array with random standard normal numbers
  Normal_random_matrix( m_A, n_A, A_pg, ldim_A, state, rs_pt );
 
  // copy matrix so we can check error later
  hipMemcpy( A_pgc, A_pg, m_A * n_A * sizeof( double ), hipMemcpyDeviceToDevice );

  // check
  hipMemcpy( A_pc, A_pg, m_A * n_A * sizeof( double ), hipMemcpyDeviceToHost );
  //print_double_matrix( A_name, m_A, n_A, A_pc, ldim_A );
  
  // compute norm to check relative error later
  for ( i=0; i < m_A * n_A; i++ ) {
    A_norm += pow( A_pc[ i ], 2 );
  }
  A_norm = sqrt( A_norm );

  // compute SVD factorization
  local_magsvd( m_A, n_A, A_pg, ldim_A,
				ss_pg,
				U_pg, ldim_U,
				Vt_pg, ldim_Vt );
  
  // compute U*D*Vt to check factorization
  gpu_dgemm( n, n, d_one, 
				m_U, n_U, U_pg, ldim_U,
				m_A, n_A, A_pg, ldim_A,
				d_zero,
				m_A, n_A, A_pg, ldim_A );
  
  gpu_dgemm( n, n, d_one, 
				m_A, n_A, A_pg, ldim_A,
				m_Vt, n_Vt, Vt_pg, ldim_Vt,
				d_neg_one,
				m_A, n_A, A_pgc, ldim_A );


  // copy result to host
  hipMemcpy( A_pc, A_pgc, m_A * n_A * sizeof( double ), hipMemcpyDeviceToHost );

  // compute relative error
  for ( i=0; i < m_A * n_A; i++ ) {
    err += pow( A_pc[ i ], 2 );
  }
  err = sqrt( err );
  

  // print out result
  //print_double_matrix( A_name, m_A, n_A, A_pc, ldim_A );
  printf( "%% ||A - UDV'||_F / ||A||_F = %e \n", err );

  // finalize magma
  magma_finalize();

  // free memory
  free( A_pc );
  hipFree( A_pg );

  hipFree( ss_pg );
  hipFree( U_pg );
  hipFree( Vt_pg );

  return 0;
}
