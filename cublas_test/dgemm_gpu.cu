
#include <hip/hip_runtime.h>
#include <hipblas.h>

void dgemm_gpu ( char transA, char transB, int m, int n, int k,
				double * alpha, double * A_pg, int ldim_A,
				double * B_pg, int ldim_B,
				double * beta, double * C_pg, int ldim_C ) {


  // generate the correct transpose option identifier that CUBLAS accepts
  hipblasOperation_t cutransA, cutransB;

  if ( transA == 'N' ) { cutransA = HIPBLAS_OP_N; }
  else if ( transA == 'T' ) { cutransA = HIPBLAS_OP_T; }

  if ( transB == 'N' ) { cutransB = HIPBLAS_OP_N; }
  else if ( transB == 'T' ) { cutransB = HIPBLAS_OP_T; }

  // create a handle for CUBLAS
  hipblasHandle_t handle;
  hipblasCreate( & handle );

  // do the multiplication
  hipblasDgemm( handle, cutransA, cutransB, m, n, k, alpha,
				A_pg, ldim_A, B_pg, ldim_B, 
				beta, C_pg, ldim_C );

  // destroy the handle
  hipblasDestroy( handle );

}

