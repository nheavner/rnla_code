#include <stdio.h>
#include <stdlib.h>
#include <hipblas.h>
#include "dgemm_gpu.h"

// =========================================================================// Declaration of local prototypes

static void print_double_matrix ( const char * name, int m_A, int n_A,
				double * buff_A, int ldim_A );

// =========================================================================
int main() {

  // some initializations
  int i;
  double d_one = 1.0, d_zero = 0.0;
  char n = 'N', t = 'T';

  // Allocate 3 arrays on CPU
  int m_A, n_A, ldim_A, m_B, n_B, ldim_B, m_C, n_C, ldim_C;

  m_A = 4; n_A = 3; ldim_A = m_A; 
  m_B = 4; n_B = 5; ldim_B = m_B;
  m_C = n_A; n_C = n_B; ldim_C = n_A;
  const char * A_name = "A";
  const char * B_name = "B";
  const char * C_name = "C";

  double * buff_A = ( double * ) malloc( m_A * n_A * sizeof( double ) );
  double * buff_B = ( double * ) malloc( m_B * n_B * sizeof( double ) );
  double * buff_C = ( double * ) malloc( m_C * n_C * sizeof( double ) );
  
  // Allocate 3 arrays on GPU
  double * buff_g_A, * buff_g_B, * buff_g_C;
  hipMalloc( & buff_g_A, m_A * n_A * sizeof( double ) );
  hipMalloc( & buff_g_B, m_B * n_B * sizeof( double ) );
  hipMalloc( & buff_g_C, m_C * n_C * sizeof( double ) );

  // Initialize matrices A,B
  for ( i=0; i<m_A*n_A; i++ ) {
    buff_A[ i ] = ( double ) i;
  }

  for ( i=0; i<m_B*n_B; i++ ) {
    buff_B[ i ] = ( double ) i;
  }

  // print matrices A,B
  print_double_matrix( A_name, m_A, n_A, buff_A, ldim_A );
  print_double_matrix( B_name, m_B, n_B, buff_B, ldim_B );

  // transfer host arrays to device (gpu)
  hipMemcpy( buff_g_A, buff_A, m_A * n_A * sizeof( double ), hipMemcpyHostToDevice );
  hipMemcpy( buff_g_B, buff_B, m_B * n_B * sizeof( double ), hipMemcpyHostToDevice );

  // do the multiplication
  dgemm_gpu( t, n, n_A, n_B, m_B,
				& d_one, buff_g_A, ldim_A,
				buff_g_B, ldim_B,
				& d_zero, buff_g_C, ldim_C );

  // copy and print result on host memory
  hipMemcpy( buff_C, buff_g_C, m_C * n_C * sizeof( double ), hipMemcpyDeviceToHost );
  print_double_matrix( C_name, m_C, n_C, buff_C, ldim_C );

  // Free GPU memory
  hipFree( buff_g_A );
  hipFree( buff_g_B );
  hipFree( buff_g_C );
  
  // Free CPU memory
  free( buff_A );
  free( buff_B );
  free( buff_C );

  return 0;

}

// =========================================================================
static void print_double_matrix(const char * name, int m_A, int n_A, 
                double * buff_A, int ldim_A ) {
  int  i, j;

  printf( "%s = [\n", name );
  for( i = 0; i < m_A; i++ ) {
    for( j = 0; j < n_A; j++ ) {
      printf( "%le ", buff_A[ i + j * ldim_A ] );
    }
    printf( "\n" );
  }
  printf( "];\n" );
}
